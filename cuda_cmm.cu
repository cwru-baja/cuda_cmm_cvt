#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <math.h>
#include <iostream>
#include <fstream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define PYBIND11

#ifdef PYBIND11
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <pybind11/stl.h>
#endif

#define ROMBERG_CONVERGENCE 1e-6f
#define ROMBERG_MAX_ITERATIONS 10
#define SEC_STEPS 50
#define SEC_TOL 1e-7f

#define FR_TOL 1e-6f
#define FR_STEPS 100

#define KAPPA_DIVISIONS 60000

#ifdef PYBIND11
#define PROGRESS_BAR if (index == blockDim.x * gridDim.x - 1) \
        { \
            const int num_chars = 20; \
            char a[num_chars + 1]; \
            a[num_chars] = '\0'; \
            for (int j = 0; j < num_chars; j++) \
            { \
                if (j * (n / num_chars) < i) \
                { \
                    a[j] = '#'; \
                } \
                else \
                { \
                    a[j] = ' '; \
                } \
            } \
            printf("Sample Thread current progress [%s]  %10d / %10d\r", a, i, n); \
        }
#else
#define PROGRESS_BAR //;
#endif

class CMM_Sheave {
    protected:
        bool converged, bisection;

        int debug;

        double kappa_array[KAPPA_DIVISIONS][3];
    public:
        float beta_naught, mu, alpha, v_theta_naught, A, delta, theta_c;
        __device__
        bool testSheaveExpansionCenter(float theta_c_naught);
        __device__
        void sheaveExpansionCenter();
        __device__
        float secNumerator(float theta);
        __device__
        float secDenominator(float theta);
        __device__
        void kappaFuncInit();
        __device__
        double rhs(float theta);
        __device__
        double kappaSlope(float theta);

        __device__
        float computeThetaC();

        // Manual Testing Constructor
        __device__
        CMM_Sheave(float beta_naught, float mu, float alpha, float v_theta_naught, float A, float delta, int debug, bool manual)
        : beta_naught{beta_naught}
        , mu{mu}
        , alpha{alpha}
        , v_theta_naught{v_theta_naught}
        , A{A}
        , delta{delta}
        , debug{debug}
        {
            this->converged = true;
            this->theta_c = 0.5752 * this->alpha;
            if (!manual)
            {
                this->sheaveExpansionCenter();
            }
        };

        __device__
        static void frBasedSheave(CMM_Sheave *sheave, float beta_naught, float mu, float alpha, float fr, float A, float delta, int debug);

        // Basic Constructor
        __device__
        CMM_Sheave(float beta_naught, float mu, float alpha, float v_theta_naught, float A, float delta, int debug)
        : CMM_Sheave(beta_naught, mu, alpha, v_theta_naught, A, delta, debug, false)
        {};

        // Non-Debug Constructor
        __device__
        CMM_Sheave(float beta_naught, float mu, float alpha, float v_theta_naught, float A, float delta)
        : CMM_Sheave(beta_naught, mu, alpha, v_theta_naught, A, delta, false)
        {};

        __device__
        CMM_Sheave()
        {};

        __device__
        float psi(float theta);
        __device__
        double kappa(float theta);
        __device__
        float press(float theta);
        __device__
        float vRadial(float theta);
        __device__
        float vTangent(float theta);
        __device__
        float beta(float theta);
        __device__
        float betaS(float theta);

        __device__
        float dimlessClamp();
        __device__
        float forceRatio();


        __device__
        float getThetaC() { return this->theta_c; };

        __device__
        static float kappaToTension(float kappa, float f_naught, float sigma, float omega, float R)
        {
            float inert = sigma * pow(omega, 2) * pow(R, 2);
            float tense = kappa * (f_naught - inert) + inert;
            return tense;
        }

        __device__
        static float sToAxialClamp(float s, float f_naught, float sigma, float omega, float R)
        {
            float inert = sigma * pow(omega, 2) * pow(R, 2);
            float axial = s * (f_naught - inert);
            return axial;
        }

        __device__
        static float axialClampToS(float axial, float f_naught, float sigma, float omega, float R)
        {
            float inert = sigma * pow(omega, 2) * pow(R, 2);
            float s = axial / (f_naught - inert);
            return s;
        }

        __device__
        static float cToTorque(float c, float f_naught, float sigma, float omega, float R)
        {
            float inert = sigma * pow(omega, 2) * pow(R, 2);
            float torque = c * R * (f_naught - inert);
            return torque;
        }

        __device__
        static float torqueToC(float torque, float f_naught, float sigma, float omega, float R)
        {
            float inert = sigma * pow(omega, 2) * pow(R, 2);
            float c = torque / (R * (f_naught - inert));
            return c;
        }

        __device__
        static float beltTensionsToForceRatio(float f_one, float f_two, float sigma, float omega, float R)
        {
            float inert = sigma * pow(omega, 2) * pow(R, 2);
            float force_ratio = (f_two - inert) / (f_one - inert);
            return force_ratio;
        }

        __device__
        static float dLdRp(float r_primary, float r_secondary, float d)
        {
            float del = M_PI + 2 * atan((r_primary - r_secondary) / d);
            del -= 2/d * (r_secondary - r_primary) / (1 + pow(((r_secondary - r_primary)/d), 2));
            del -= 2 * r_primary / sqrt(pow(d, 2) + pow((r_secondary - r_primary), 2));

            return del;
        }

        __device__
        static float dLdRs(float r_primary, float r_secondary, float d)
        {
            return CMM_Sheave::dLdRp(r_secondary, r_primary, d);
        }
};


__device__ float secNumeratorEval(float theta, CMM_Sheave * sheave)
{
    return sheave->press(theta) * sin(theta);
}

__device__ float secDenominatorEval(float theta, CMM_Sheave * sheave)
{
    return sheave->press(theta) * cos(theta);
}

__device__ float dimlessClampEval(float theta, CMM_Sheave * sheave)
{
    return (cos(sheave->beta(theta)) + sheave->mu * sin(sheave->betaS(theta))) * sheave->press(theta);
}

__device__ float trapSum(float (*f)(float, CMM_Sheave*), CMM_Sheave *sheave, float start, float end, int n) {
    float h = (end - start) / n;

    float sum = 0.5f * (f(start, sheave) + f(end, sheave));

    for (int i = 1; i < n; i++) {
        sum += f(start + i * h, sheave);
    }

    return sum * h;
}

__device__ float rombergSum(float (*f)(float, CMM_Sheave*), CMM_Sheave *sheave, float start, float end) {
    float convergence = 1e9;

    float romberg[ROMBERG_MAX_ITERATIONS - 1];
    float last_romberg[ROMBERG_MAX_ITERATIONS - 2];
    int n = 1;
    last_romberg[0] = trapSum(f, sheave, start, end, n);

    int iter = 2;


    while (convergence > ROMBERG_CONVERGENCE && iter < ROMBERG_MAX_ITERATIONS) {
        n *= 2;

        romberg[0] = trapSum(f, sheave, start, end, n);

        for (int k = 1; k < iter; k++) {
            romberg[k] = (pow(4, k) * romberg[k - 1] - last_romberg[k - 1]) / (pow(4, k) - 1);
        }

        convergence = abs((romberg[iter - 1] - last_romberg[iter - 2]) / romberg[iter - 1]);
        // printf("Romberg Iter: %d, Convergence: %f, Value: %f\n", iter, convergence, romberg[iter - 1]);
        iter++;

        for (int i = 0; i < iter - 1; i++) {
            last_romberg[i] = romberg[i];
        }
    }

    // printf("Romberg Iter: %d, Convergence: %f, Value: %f\r", iter, convergence, romberg[iter - 1]);
    return last_romberg[iter - 2];
}

// Takes in log(fr) and inits a sheave that is close enought to that force ratio
// Modified Secant Method
__device__
void CMM_Sheave::frBasedSheave(CMM_Sheave *sheave, float beta_naught, float mu, float alpha, float fr, float A, float delta, int debug)
{
    if (debug)
        printf("FR Target: %f\n", fr);

    sheave->beta_naught = beta_naught;
    sheave->mu = mu;
    sheave->alpha = alpha;
    sheave->A = A;
    sheave->delta = delta;
    sheave->debug = (debug > 2) * (debug - 2);

    float last_vel = 0.0, vel = 0.2, mid;

    sheave->v_theta_naught = vel;
    sheave->sheaveExpansionCenter();
    float current_fr = sheave->forceRatio();

    int iter = 0;
    last_vel = -5e3;
    vel = 5e3;

    if (debug > 1) {
        sheave->v_theta_naught = last_vel;
        sheave->sheaveExpansionCenter();

        printf("Lower Bound FR: %f\n", sheave->forceRatio());

        sheave->v_theta_naught = vel;
        sheave->sheaveExpansionCenter();

        printf("Upper Bound FR: %f\n", sheave->forceRatio());
     }

    while (iter < FR_STEPS && vel - last_vel > 1e-5)
    {
        mid = (last_vel + vel) / 2;
        sheave->v_theta_naught = mid;
        sheave->sheaveExpansionCenter();

        current_fr = sheave->forceRatio();

        if (debug > 1)
            printf("Mid Vel: %f, Current FR: %f\n", mid, current_fr);

        if (current_fr < fr)
            last_vel = mid;
        else
            vel = mid;

        if (debug > 1)
            printf("FR Iter: %d Vel Bounds: [%f, %f]\n", iter, last_vel, vel);
        iter++;
    }

    if (debug)
        printf("FR Iterations: %d, Target FR: %f, Final FR: %f, Vel: %f\n", iter, fr, current_fr, vel);
}

__device__
float CMM_Sheave::computeThetaC()
{
    float y = rombergSum(secNumeratorEval, this, 0, this->alpha);
    float x = rombergSum(secDenominatorEval, this, 0, this->alpha);

    return atan2(y, x);
}

__device__
void CMM_Sheave::sheaveExpansionCenter()
{
    int iter = 0;

    double lower_tc, upper_tc;

    this->kappaFuncInit();
    float theta_c_prime = this->computeThetaC();

    if (theta_c_prime < this->theta_c) {
        lower_tc = 0.0;
        upper_tc = this->theta_c;
    } else {
        lower_tc = this->theta_c;
        upper_tc = this->alpha;
    }


    while (upper_tc - lower_tc > SEC_TOL && iter < SEC_STEPS)
    {
        this->theta_c = (upper_tc + lower_tc) / 2;

        this->kappaFuncInit();

        theta_c_prime = this->computeThetaC();

        if (theta_c_prime > this->theta_c)
            lower_tc = this->theta_c;
        else
            upper_tc = this->theta_c;

        iter++;

        if (this->debug > 1)
        {
            printf("Iteration %d: (%f, %f)\n", iter, lower_tc, upper_tc);
        }
    }

    // printf("SEC Iterations: %d\n", iter);
    // this->converged = iter < SEC_STEPS;
    this->converged = true;
}

__device__
void CMM_Sheave::kappaFuncInit()
{
    // Kappa is euler method nearest neighbor (linear interp between computed points)
    // this->kappa_array[ind] = [theta, rhs, rhs_slope]

    double rhs_current = 0.0;
    double rhs_delta = 0.0;

    double start_slope = 0.0;
    double end_slope = 0.0;
    double avg_slope = 0.0;

    double t = 0.0;

    // Resolution of Kappa in the Theta domain
    double kappa_res = this->alpha / (KAPPA_DIVISIONS - 1.0);

    for (int ind = 0; ind < KAPPA_DIVISIONS; ind++)
    {
        t = ind * kappa_res;

        start_slope = this->kappaSlope(t);

        // Predictor Step
        rhs_delta = kappa_res * start_slope;
        end_slope = this->kappaSlope(t + kappa_res);

        // Corrector Step
        avg_slope = 0.5 * (start_slope + end_slope);
        rhs_delta = avg_slope * kappa_res;

        this->kappa_array[ind][0] = t;
        this->kappa_array[ind][1] = rhs_current;
        this->kappa_array[ind][2] = avg_slope;

        rhs_current += rhs_delta;
    }
}

__device__
double CMM_Sheave::kappaSlope(float theta)
{
    float vRad = this->vRadial(theta);
    float vTan = this->vTangent(theta);
    double vMag = sqrt(vRad * vRad + vTan * vTan);

    double cos_p = vRad / vMag;
    return this->mu * (vTan / vMag) / (sin(this->beta_naught) * sqrt(1 + pow(tan(this->beta_naught)*cos_p, 2)) - this->mu * cos_p);
}

__device__
double CMM_Sheave::rhs(float theta)
{
    int ind = (int) (theta / (this->alpha / (KAPPA_DIVISIONS - 1)));

    double* k = this->kappa_array[ind];

    // std::cout << "RHS DEBUG: " << k[1] << " " << k[2] << " " << theta << " " << k[0] << std::endl;

    return k[1] + k[2] * (theta - k[0]);
}

__device__
double CMM_Sheave::kappa(float theta)
{
    // if (theta - this->alpha > 0.1)
    // {
    //     std::cout << "Error: theta > alpha (" << theta << ", " << this->alpha << ")" << std::endl;
    // }
    // else if (theta < 0.0)
    // {
    //     std::cout << "Error: theta < 0 (" << theta << ")" << std::endl;
    // }

    int ind = (int) (theta / (this->alpha / (KAPPA_DIVISIONS - 1)));

    double* k = this->kappa_array[ind];

    double rhs_theta = k[1] + k[2] * (theta - k[0]);

    return exp(rhs_theta);
}

__device__
float CMM_Sheave::psi(float theta)
{
    float vRad = this->v_theta_naught - this->A*theta - 2*sin(theta/2)*sin(theta/2 - this->theta_c);
    float vTan = this->A - cos(theta - this->theta_c + 0.5*M_PI);

    return atan2(vRad, vTan);
}

// EQ. 37
__device__
float CMM_Sheave::press(float theta)
{
    float vRad = this->vRadial(theta);
    float vTan = this->vTangent(theta);

    float cos_p = vRad / sqrt(vRad * vRad + vTan * vTan);

    float tan_2_beta = pow(tan(this->beta_naught), 2);

    return sqrt(1 + tan_2_beta * pow(cos_p, 2)) / (sin(this->beta_naught) * sqrt(1 + tan_2_beta * pow(cos_p, 2)) - this->mu * cos_p) * this->kappa(theta) * 0.5;
}

// EQ. 35
__device__
float CMM_Sheave::vRadial(float theta)
{
    return this->A - cos(theta - this->theta_c + 0.5*M_PI);
}

// EQ. 36
__device__
float CMM_Sheave::vTangent(float theta)
{
    return this->v_theta_naught - this->A*theta - 2*sin(theta/2)*sin(theta/2 - this->theta_c);
}

// EQ. 17
__device__
float CMM_Sheave::beta(float theta)
{
    return this->beta_naught + 0.5 * this->delta * sin(theta - this->theta_c + 0.5 * M_PI);
}

// EQ. 4
__device__
float CMM_Sheave::betaS(float theta)
{
    float vRad = this->vRadial(theta);
    float vTan = this->vTangent(theta);

    float cos_p = vRad / sqrt(vRad * vRad + vTan * vTan);

    return atan(tan(this->beta(theta)) * cos_p);
}

// EQ. 50
__device__
float CMM_Sheave::dimlessClamp()
{
    return rombergSum(dimlessClampEval, this, 0.0, this->alpha);
}

__device__
float CMM_Sheave::forceRatio()
{
    if (this->converged)
    {
        return this->rhs(this->alpha);
    }
    else
    {
        return nanf("e");
    }
}

__device__
float computeL(float primary, float tau, float cToC)
{
    float secondary = primary / tau;
    float theta_t = asinf((secondary - primary) / cToC);

    float alpha_primary = M_PI - 2 * theta_t;
    float alpha_secondary = M_PI + 2 * theta_t;
    return primary * alpha_primary + secondary * alpha_secondary + 2 * sqrtf(cToC*cToC - (secondary - primary) * (secondary - primary));
}

__device__
float lSlope(float primary, float tau, float cToC)
{
    float secondary = primary / tau;
    float theta_t = asinf((secondary - primary) / cToC);

    float alpha_primary = M_PI - 2 * theta_t;
    float alpha_secondary = M_PI + 2 * theta_t;

    float dratio_dp = (1/tau - 1) / cToC;

    float dtt_dp = 1.0f / sqrtf(1 + powf(((primary * (1/tau - 1)) / cToC), 2)) * dratio_dp;

    float dap_dp = -2.0f * dtt_dp;
    float das_dp = 2.0f * dtt_dp;

    float dsl_dp = 0.5f / sqrtf(cToC*cToC - powf((secondary - primary), 2)) * 2.0f * primary * (1/tau - 1) * (1/tau - 1);

    return alpha_primary + primary * dap_dp + 1.0f / tau * alpha_secondary + secondary * das_dp + 2 * dsl_dp;
}

__device__
float tauToPrimary(float tau, float cToC, float L)
{
    float guess = (L - 2 * cToC) / (1 + 1/tau) / M_PI, length;

    int iter = 0;
    float discrep = 1e9;

    while (fabs(discrep) > 1e-6 && iter < 50)
    {
        length = computeL(guess, tau, cToC);
        discrep = length - L;

        guess -= discrep / lSlope(guess, tau, cToC);
        iter++;
    }

    return guess;
}

__device__
float dLdRp(float r_primary, float r_secondary, float d)
{
    float del = M_PI + 2 * atan((r_primary - r_secondary) / d);
    del -= 2/d * (r_secondary - r_primary) / (1 + pow(((r_secondary - r_primary)/d), 2));
    del -= 2 * r_primary / sqrt(pow(d, 2) + pow((r_secondary - r_primary), 2));

    return del;
}

__device__
float dLdRs(float r_primary, float r_secondary, float d)
{
    return dLdRp(r_secondary, r_primary, d);
}

__global__
void fr(int n, float *vec, float beta_naught, float mu)
{
    const int cols = 4;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride) {
        PROGRESS_BAR

        float v_theta_naught = vec[cols * i + 0];
        float A = vec[cols * i + 1];
        float alpha = vec[cols * i + 2];

        CMM_Sheave sheave(beta_naught, mu, alpha, v_theta_naught, A, 0.0f, false, false);

        vec[cols * i + 3] = sheave.forceRatio();
    }
}

__global__
void equilibriumClamp(int n, float *vec, float beta_naught, float mu, float delta)
{
    const int cols = 3;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride) {
        PROGRESS_BAR

        float fr = vec[cols * i + 0];
        float alpha = vec[cols * i + 1];

        CMM_Sheave sheave;
        CMM_Sheave::frBasedSheave(&sheave, beta_naught, mu, alpha, fr, 0.0, delta, 0);

        // if (fabsf(sheave.forceRatio() - fr) > 0.2f) {
        //     printf("Fr In: %f alpha: %f Fr Out:%f\n", fr, alpha, sheave.forceRatio());
        // }

        vec[cols * i + 0] = sheave.forceRatio();
        vec[cols * i + 2] = sheave.dimlessClamp();
    }
}

__global__
void cCoefficientCompute(int n, float *vec, float beta_naught, float mu, float delta, float cToC, float L, float sigma)
{
    const int cols = 6;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride) {
        PROGRESS_BAR

        float fr = vec[cols * i + 0];
        float A = vec[cols * i + 1];
        float tau = vec[cols * i + 2];

        // printf("\nfr %f A %f tau %f omega %f\n", fr, A, tau, omega);

        float prim_rad = tauToPrimary(tau, cToC, L);
        // printf("Primary Radius: %f\n", prim_rad);

        float theta_t = asinf(((1 / tau - 1) * prim_rad) / cToC);

        float alpha_primary = M_PI - 2 * theta_t;
        float alpha_secondary = M_PI + 2 * theta_t;
        // printf("Alpha Primary: %f Alpha Secondary: %f\n", alpha_primary, alpha_secondary);

        // float taught_side_force =

        // printf("Primary FR: %f Secondary FR: %f\n", fr, -fr);
        float secondary_A = -A * dLdRp(prim_rad, prim_rad / tau, cToC) / dLdRs(prim_rad, prim_rad / -fr, cToC);
        // printf("Primary A: %f Secondary A: %f\n", A, secondary_A);

        CMM_Sheave sheave;
        CMM_Sheave::frBasedSheave(&sheave, beta_naught, mu, alpha_primary, fr, 0.0, delta, 0);
        float prim_eq = sheave.dimlessClamp();

        fr = sheave.forceRatio();

        CMM_Sheave::frBasedSheave(&sheave, beta_naught, mu, alpha_secondary, -fr, 0.0, delta, 0);
        float sec_eq = sheave.dimlessClamp();

        CMM_Sheave::frBasedSheave(&sheave, beta_naught, mu, alpha_primary, fr, A, delta, 0);
        float prim_clamp = sheave.dimlessClamp();

        float prim_v_theta = sheave.vTangent(0.0);
        float prim_v_r = sheave.vRadial(0.0);

        float D = sinf(2 * beta_naught) / (delta * (1 + powf(cosf(beta_naught), 2)));


        CMM_Sheave::frBasedSheave(&sheave, beta_naught, mu, alpha_secondary, -fr, secondary_A, delta, 0);
        float sec_clamp = sheave.dimlessClamp();
        float sec_v_theta = sheave.vTangent(alpha_secondary);
        float sec_v_r = sheave.vRadial(alpha_secondary);

        float tau_effective =  tau * (1 - prim_v_theta / (prim_v_theta - D)) / (1 - sec_v_theta / (sec_v_theta - D)) * sqrtf((1 + prim_v_r*prim_v_r / D / D) / (1 + sec_v_r*sec_v_r / D / D));

        vec[cols * i + 0] = fr;

        vec[cols * i + 3] = logf(prim_clamp / sec_clamp * sec_eq / prim_eq);
        vec[cols * i + 4] = tau_effective;

        vec[cols * i + 5] = prim_clamp;
    }
}

#ifdef PYBIND11
// Take in a n x 4 matrix and write the force ratio to the final column
void pyFR(pybind11::array_t<float> vec, float beta_naught, float mu)
{
    const int cols = 4;

    pybind11::buffer_info ha = vec.request();

    if (ha.ndim != 2 && ha.shape[1] != cols) {
        std::stringstream strstr;
        strstr << "Numpy Array not n x " << cols << std::endl;
        throw std::runtime_error(strstr.str());
    }

    int n = ha.shape[0];
    size_t size_bytes = n*cols*sizeof(float);
    float *gpu_ptr;
    hipError_t error;

    error = hipMalloc(&gpu_ptr, size_bytes);

    if (error != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(error));
    }

    float* ptr = reinterpret_cast<float*>(ha.ptr);
    error = hipMemcpy(gpu_ptr, ptr, size_bytes, hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(error));
    }

    int deviceID;
    hipDeviceProp_t props;

    hipGetDevice(&deviceID);
    hipGetDeviceProperties(&props, deviceID);

    int num_blocks = props.multiProcessorCount;
    int num_threads = 256;

    fr<<<num_blocks, num_threads>>>(n, gpu_ptr, beta_naught, mu);

    error = hipGetLastError();
    if (error != hipSuccess) {
        std::stringstream strstr;
        strstr << "run_kernel launch failed" << std::endl;
        strstr << hipGetErrorString(error);
        throw strstr.str();
    }

    error = hipMemcpy(ptr, gpu_ptr, size_bytes, hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(error));
    }

    error = hipFree(gpu_ptr);
    if (error != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(error));
    }
}

// Take in a n x 3 matrix and write the dimensionless clamp to the final column
void pyEquilibrium(pybind11::array_t<float> vec, float beta_naught, float mu, float delta)
{
    const int cols = 3;
    pybind11::buffer_info ha = vec.request();

    if (ha.ndim != 2 && ha.shape[1] != cols) {
        std::stringstream strstr;
        strstr << "Numpy Array not n x " << cols << std::endl;
        throw std::runtime_error(strstr.str());
    }

    int n = ha.shape[0];
    size_t size_bytes = n*cols*sizeof(float);
    float *gpu_ptr;
    hipError_t error;

    error = hipMalloc(&gpu_ptr, size_bytes);

    if (error != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(error));
    }

    float* ptr = reinterpret_cast<float*>(ha.ptr);
    error = hipMemcpy(gpu_ptr, ptr, size_bytes, hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(error));
    }

    int deviceID;
    hipDeviceProp_t props;

    hipGetDevice(&deviceID);
    hipGetDeviceProperties(&props, deviceID);

    int num_blocks = props.multiProcessorCount;
    int num_threads = 256;

    equilibriumClamp<<<num_blocks, num_threads>>>(n, gpu_ptr, beta_naught, mu, delta);

    error = hipGetLastError();
    if (error != hipSuccess) {
        std::stringstream strstr;
        strstr << "run_kernel launch failed" << std::endl;
        strstr << hipGetErrorString(error);
        throw strstr.str();
    }

    error = hipMemcpy(ptr, gpu_ptr, size_bytes, hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(error));
    }

    error = hipFree(gpu_ptr);
    if (error != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(error));
    }
}

// Take in a n x 6 matrix and write the dimensionless clamp ratio delta and the tau effective to the final columns
void pyCCoefficient(pybind11::array_t<float> vec, float beta_naught, float mu, float delta, float cToC, float L, float sigma)
{
    const int cols = 6;

    pybind11::buffer_info ha = vec.request();

    if (ha.ndim != 2 && ha.shape[1] != cols) {
        std::stringstream strstr;
        strstr << "Numpy Array not n x " << cols << std::endl;
        throw std::runtime_error(strstr.str());
    }

    int n = ha.shape[0];
    size_t size_bytes = n*cols*sizeof(float);
    float *gpu_ptr;
    hipError_t error;

    error = hipMalloc(&gpu_ptr, size_bytes);

    if (error != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(error));
    }

    float* ptr = reinterpret_cast<float*>(ha.ptr);
    error = hipMemcpy(gpu_ptr, ptr, size_bytes, hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(error));
    }

    int deviceID;
    hipDeviceProp_t props;

    hipGetDevice(&deviceID);
    hipGetDeviceProperties(&props, deviceID);

    int num_blocks = props.multiProcessorCount;
    int num_threads = 256;

    cCoefficientCompute<<<num_blocks, num_threads>>>(n, gpu_ptr, beta_naught, mu, delta, cToC, L, sigma);

    error = hipGetLastError();
    if (error != hipSuccess) {
        std::stringstream strstr;
        strstr << "run_kernel launch failed" << std::endl;
        strstr << hipGetErrorString(error);
        throw strstr.str();
    }

    error = hipMemcpy(ptr, gpu_ptr, size_bytes, hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(error));
    }

    error = hipFree(gpu_ptr);
    if (error != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(error));
    }
}

PYBIND11_MODULE(cuda_cmm, m)
{
    m.def("fr", &pyFR);
    m.def("eq_clamp", &pyEquilibrium);
    m.def("c_coefficient", &pyCCoefficient);
}
#endif

#ifndef PYBIND11
int main() {
    const size_t n = 8*64;
    float a[n] = {-3.600783, -0.935484, 3.695991, -69.420, -3.491194, -0.935484, 3.563651, -69.420};

    float *gpu_ptr;
    hipMalloc(&gpu_ptr, n*sizeof(float));

    hipMemcpy(gpu_ptr, a, n*sizeof(float), hipMemcpyHostToDevice);

    fr<<<2, 1>>>(n/4, gpu_ptr, M_PI / 180.0 * 23.0 / 2.0, 0.1);

    hipDeviceSynchronize();
    hipMemcpy(a, gpu_ptr, n*sizeof(float), hipMemcpyDeviceToHost);

    std::cout << a[0] << " " << a[1] << " " << a[2] << " " << a[3] << std::endl;
    std::cout << a[4] << " " << a[5] << " " << a[6] << " " << a[7] << std::endl;

    hipFree(gpu_ptr);


    /* float b[3] = {-1.4, 2.8274333, 0.0};
    hipMalloc(&gpu_ptr, 3*sizeof(float));

    hipMemcpy(gpu_ptr, b, 3*sizeof(float), hipMemcpyHostToDevice);

    equilibriumClamp<<<1, 1>>>(1, gpu_ptr, M_PI / 180.0 * 23.0 / 2.0, 0.1, 1e-4);

    hipDeviceSynchronize();
    hipMemcpy(b, gpu_ptr, 3*sizeof(float), hipMemcpyDeviceToHost);

    std::cout << b[0] << " " << b[1] << " " << b[2] << std::endl;

    hipFree(gpu_ptr);

    const int ps = 12;
    float c[ps] = {-1.4, 0.5, 0.8, 0.0, -1.0, 0.5, 0.8, 0.0, -0.4, 0.5, 0.8, 0.0};
    hipMalloc(&gpu_ptr, ps*sizeof(float));

    hipMemcpy(gpu_ptr, c, ps*sizeof(float), hipMemcpyHostToDevice);

    cCoefficientCompute<<<1, 1>>>(3, gpu_ptr, M_PI / 180.0 * 23.0 / 2.0, 0.1, 1e-4, 0.3, 2.0, 0.5);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(error));
    }

    hipDeviceSynchronize();
    hipMemcpy(c, gpu_ptr, ps*sizeof(float), hipMemcpyDeviceToHost);

    std::cout << c[0] << " " << c[1] << " " << c[2] << " " << c[3] << " " << c[4] << std::endl;


    hipFree(gpu_ptr); */

    return 0;
}
#endif
